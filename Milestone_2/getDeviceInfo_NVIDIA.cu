#include <fstream>
#include <iostream>
using namespace std;

// compile this with: nvcc getDeviceInfo.cu -o getDeviceInfo

int CUDA_CHECK(hipError_t status) {
  if (status != hipSuccess) {
    printf("FAIL: call='%s'. Reason:%s\n", #call, hipGetErrorString(status));
    exit(EXIT_FAILURE);
  }
}

int main() {
  cout << "Querying NVIDIA device info...\n\n";

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    cout << "hipGetDeviceCount returned " << static_cast<int>(error_id)
         << "\n-> " << hipGetErrorString(error_id) << "\nResult = FAIL\n";
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (0 == deviceCount) {
    cout << "There are no available device(s) that support CUDA\n";
  } else {
    cout << "Detected " << deviceCount << " CUDA Capable device(s)\n";
  }

  int deviceId = 0;
  CUDA_CHECK(hipGetDevice(&deviceId));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceId);

  int driverVersion = 0, runtimeVersion = 0;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  cout << "Info for device " << deviceId << "\n  === Device attributes === "
       << "\n  | Device Name: " << deviceProp.name
       << "\n  | CUDA Driver Version / Runtime Version: "
       << driverVersion / 1000 << "." << (driverVersion % 100) / 10 << " / "
       << runtimeVersion / 1000 << "." << (runtimeVersion % 100) / 10
       << "\n  | CUDA Capability Major/Minor version number: "
       << deviceProp.major << "." << deviceProp.minor
       << "\n  | Number of SMs: " << deviceProp.multiProcessorCount
       << "\n  | Threads per Warp: " << deviceProp.warpSize
       << "\n  === SM attributes === "
       << "\n  | Maximum Number of Threads per SM: "
       << deviceProp.maxThreadsPerMultiProcessor
       << "\n  === Threadblock attributes === "
       << "\n  | Maximum Shared Memory per Block: "
       << deviceProp.sharedMemPerBlock
       << "\n  | Maximum Number of Threads per Block: "
       << deviceProp.maxThreadsPerBlock
       << "\n  | Maximum Number of Registers per Block: "
       << deviceProp.regsPerBlock << "\n  ===" << endl;

  // to get the number of cores per SM, we need to know the major & minor
  // version of the GPU
  // https://stackoverflow.com/questions/32530604/how-can-i-get-number-of-cores-in-cuda-device
  // https://github.com/NVIDIA/cuda-samples/blob/6be514679b201c8a0f0cda050bc7c01c8cda32ec/Common/hip/hip_runtime_api.h
  int compute_cores_per_SM; // = cuda cores per SM
  switch (deviceProp.major) {
  case 2: // Fermi
    if (deviceProp.minor == 1)
      compute_cores_per_SM = 48;
    else
      compute_cores_per_SM = 32;
    break;
  case 3: // Kepler
    compute_cores_per_SM = 192;
    break;
  case 5: // Maxwell
    compute_cores_per_SM = 128;
    break;
  case 6: // Pascal
    if ((deviceProp.minor == 1) || (deviceProp.minor == 2))
      compute_cores_per_SM = 128;
    else if (deviceProp.minor == 0)
      compute_cores_per_SM = 64;
    else
      cout << "Unknown device type\n";
    break;
  case 7: // Volta & Turing
    if ((deviceProp.minor == 0) || (deviceProp.minor == 5))
      compute_cores_per_SM = 64;
    else
      cout << "Unknown device type\n";
    break;
  case 8: // Ampere
    if (deviceProp.minor == 0)
      compute_cores_per_SM = 64;
    else if (deviceProp.minor == 6)
      compute_cores_per_SM = 128;
    else
      cout << "Unknown device type\n";
    break;
  default:
    cout << "Unknown device type\n";
    break;
  }

  ofstream out;
  out.open("device_data_NVIDIA.py");
  if (!out) {
    cerr << "Error: file could not be opened" << endl;
    exit(1);
  }

  int warps_per_SM = (int)(deviceProp.maxThreadsPerMultiProcessor /
                           deviceProp.maxThreadsPerBlock);
  // number of compute cores per SM = warps_per_SM * warpSize?
  int compute_cores_per_SM = warps_per_SM * deviceProp.warpSize;
  out << "Name = \"" << deviceProp.name << "\""
      << "\nSMs = " << deviceProp.multiProcessorCount
      << "\nwarps_per_SM = " << warps_per_SM
      << "\nthreads_per_warp = " << deviceProp.warpSize
      << "\nregisters_per_thread_block = " << deviceProp.regsPerBlock
      << "\nregisters_per_warp = " << deviceProp.regsPerBlock
      << "\ntotal_compute_cores = "
      << compute_cores_per_SM * deviceProp.multiProcessorCount
      << "\ncapability_version = " << deviceProp.major << "."
      << deviceProp.minor;

  out.close();
  return 0;
}