#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>

constexpr long long VECLEN = 4;
constexpr float alpha = 1;
constexpr float beta = 0;
constexpr long long size_thread_block_tile_m = 128;
constexpr long long size_thread_block_tile_n = 128;
constexpr long long size_K_tile = 8;
constexpr long long num_thread_blocks_m = 32;
constexpr long long num_thread_blocks_n = 32;
constexpr long long num_K_tiles = 512;
constexpr long long size_warp_tile_m = 32;
constexpr long long size_warp_tile_n = 64;
constexpr long long size_thread_tile_m = 8;
constexpr long long size_thread_tile_n = 8;
constexpr long long SPLIT_K = 1;
constexpr long long num_threads_per_threadblock = 256;

struct gemm_t {
    dace::cuda::Context *gpu_context;
};


#define TYPE float
#define VECTORTYPE2 float2
#define VECTORTYPE4 float4
// #define M 4096
// #define N 4096
// #define K 4096
#define THREADBLOCK_TILE_M 128
#define THREADBLOCK_TILE_N 128
#define THREADBLOCK_TILE_K 4096
#define LOAD_K 8
#define WARP_TILE_M 32
#define WARP_TILE_N 64
#define THREAD_TILE_M 8
#define THREAD_TILE_N 8
#define A_OFFSET 0
#define B_OFFSET 0
// #define SWIZZLE 1
// #define SPLIT_K 1
#define ATOMIC_REDUCTION false
#define ADDITIONAL_OCCUPANCY_SM 2
#define ALPHA 1
#define BETA 0


/**
 * This function loads the values of A from shared memory into registers.
 *
 * @param A_Shared			The shared memory to store the tile, column major
 * @param A_register		Registers to store A
 * @param k					Current k index to load
 * @param WarpIdy			The WarpId in the y dimension of the current thread
 * @param LaneIdy			The LaneId in the y dimension of the current thread
 * @param A_Shared_Offset	Offset used to access A_Shared due to double buffering
 */
 __device__ __inline__ void load_A_Shared(
    const TYPE (* __restrict__ A_Shared)[2 * (THREADBLOCK_TILE_M + A_OFFSET)
            * LOAD_K],
    TYPE (* __restrict__ A_register)[ THREAD_TILE_M], const int k,
    const int WarpIdy, const int LaneIdy, const int A_Shared_Offset) {

constexpr int TIMES = THREAD_TILE_M / 4;

constexpr int M_THREADS = WARP_TILE_M / THREAD_TILE_M;

const int Shared_j = k;

// We use as many float4 loads as we can
#pragma unroll
for (int i = 0; i < TIMES; i++) {

    const int Shared_i = WarpIdy * WARP_TILE_M + i * M_THREADS * 4
            + LaneIdy * 4;

    const TYPE* shared_mem_pointer = &(*A_Shared)[A_Shared_Offset + Shared_i
            + (THREADBLOCK_TILE_M + A_OFFSET) * Shared_j];

    const VECTORTYPE4 a =
            reinterpret_cast<const VECTORTYPE4*>(shared_mem_pointer)[0];

    TYPE* register_ptr = &(*A_register)[i * 4];

    reinterpret_cast<VECTORTYPE4*>(register_ptr)[0] = a;

}

// If there is a rest greater equal 2, we can use one more float 2 load
if (THREAD_TILE_M % 4 >= 2) {

    const int Shared_i = WarpIdy * WARP_TILE_M + TIMES * M_THREADS * 4
            + LaneIdy * 2;

    const TYPE* shared_mem_pointer = &(*A_Shared)[A_Shared_Offset + Shared_i
            + (THREADBLOCK_TILE_M + A_OFFSET) * Shared_j];

    const VECTORTYPE2 a =
            reinterpret_cast<const VECTORTYPE2*>(shared_mem_pointer)[0];

    TYPE* register_ptr = &(*A_register)[TIMES * 4];

    reinterpret_cast<VECTORTYPE2*>(register_ptr)[0] = a;

}

// And use one single load in the end, if there is still some rest
if (THREAD_TILE_M % 2 > 0) {

    constexpr int ADDITIONAL_OFFSET_SHARED =
            (THREAD_TILE_M % 4 >= 2) ? M_THREADS * 2 : 0;

    constexpr int ADDITIONAL_OFFSET_REGISTER =
            (THREAD_TILE_M % 4 >= 2) ? 2 : 0;

    const int Shared_i = WarpIdy * WARP_TILE_M + TIMES * M_THREADS * 4
            + LaneIdy + ADDITIONAL_OFFSET_SHARED;

    (*A_register)[TIMES * 4 + ADDITIONAL_OFFSET_REGISTER] =
            (*A_Shared)[A_Shared_Offset + Shared_i
                    + (THREADBLOCK_TILE_M + A_OFFSET) * Shared_j];

}

}

/**
* This function loads the values of B from shared memory into registers.
*
* @param B_Shared			The shared memory to store the tile, row major
* @param B_register		Registers to store B
* @param k					Current k index to load
* @param WarpIdx			The WarpId in the x dimension of the current thread
* @param LaneIdx			The LaneId in the x dimension of the current thread
* @param B_Shared_Offset 	Offset used to access B_Shared due to double buffering
*/
__device__ __inline__ void load_B_Shared(
TYPE (* __restrict__ B_Shared)[2 * LOAD_K * (THREADBLOCK_TILE_N + B_OFFSET)],
TYPE (* __restrict__ B_register)[ THREAD_TILE_N], const int k,
    const int WarpIdx, const int LaneIdx, const int B_Shared_Offset) {

constexpr int N_THREADS = WARP_TILE_N / THREAD_TILE_N;

constexpr int TIMES = THREAD_TILE_N / 4;

const int Shared_i = k;

// We use as many float4 loads as we can
#pragma unroll
for (int i = 0; i < TIMES; i++) {

    const int Shared_j = WarpIdx * WARP_TILE_N + LaneIdx * 4
            + i * N_THREADS * 4;

    const TYPE* shared_mem_pointer = &(*B_Shared)[B_Shared_Offset
            + Shared_i * (THREADBLOCK_TILE_N + B_OFFSET) + Shared_j];

    const VECTORTYPE4 a =
            reinterpret_cast<const VECTORTYPE4*>(shared_mem_pointer)[0];

    TYPE* register_ptr = &(*B_register)[i * 4];

    reinterpret_cast<VECTORTYPE4*>(register_ptr)[0] = a;

}

// If there is a rest greater equal 2, we can use one more float 2 load
if (THREAD_TILE_N % 4 >= 2) {

    const int Shared_j = WarpIdx * WARP_TILE_N + LaneIdx * 2
            + TIMES * N_THREADS * 4;

    const TYPE* shared_mem_pointer = &(*B_Shared)[B_Shared_Offset
            + Shared_i * (THREADBLOCK_TILE_N + B_OFFSET) + Shared_j];

    const VECTORTYPE2 a =
            reinterpret_cast<const VECTORTYPE2*>(shared_mem_pointer)[0];

    TYPE* register_ptr = &(*B_register)[TIMES * 4];

    reinterpret_cast<VECTORTYPE2*>(register_ptr)[0] = a;

}

// And use one single load in the end, if there is still some rest
if (THREAD_TILE_N % 2 > 0) {

    constexpr int ADDITIONAL_OFFSET_SHARED =
            (THREAD_TILE_N % 4 >= 2) ? N_THREADS * 2 : 0;

    constexpr int ADDITIONAL_OFFSET_REGISTER =
            (THREAD_TILE_N % 4 >= 2) ? 2 : 0;

    const int Shared_j = WarpIdx * WARP_TILE_N + LaneIdx
            + TIMES * N_THREADS * 4 + ADDITIONAL_OFFSET_SHARED;

    (*B_register)[TIMES * 4 + ADDITIONAL_OFFSET_REGISTER] =
            (*B_Shared)[B_Shared_Offset
                    + Shared_i * (THREADBLOCK_TILE_N + B_OFFSET) + Shared_j];

}

}

/**
*
* This function loads the values of A and B from shared memory into registers.
*
* @param A_Shared			The shared memory to store the tile, column major
* @param A_register		Registers to store A
* @param B_Shared			The shared memory to store the tile, row major
* @param B_register		Registers to store B
* @param k					Current k index to load
* @param WarpIdx			The WarpId in the x dimension of the current thread
* @param WarpIdy			The WarpId in the y dimension of the current thread
* @param LaneIdx			The LaneId in the x dimension of the current thread
* @param LaneIdy			The LaneId in the y dimension of the current thread
* @param A_Shared_Offset 	Offset used to access A_Shared due to double buffering
* @param B_Shared_Offset 	Offset used to access B_Shared due to double buffering
*/
__device__ __inline__ void load_Shared(
TYPE (* __restrict__ A_Shared)[2 * (THREADBLOCK_TILE_M + A_OFFSET) * LOAD_K],
TYPE (* __restrict__ A_register)[THREAD_TILE_M],
TYPE (* __restrict__ B_Shared)[2 * LOAD_K * (THREADBLOCK_TILE_N + B_OFFSET)],
TYPE (* __restrict__ B_register)[THREAD_TILE_N], const int k, const int WarpIdx,
    const int WarpIdy, const int LaneIdx, const int LaneIdy,
    const int A_Shared_Offset, const int B_Shared_Offset) {

load_A_Shared(A_Shared, A_register, k, WarpIdy, LaneIdy, A_Shared_Offset);

load_B_Shared(B_Shared, B_register, k, WarpIdx, LaneIdx, B_Shared_Offset);
}


DACE_EXPORTED int __dace_init_cuda(gemm_t *__state, int K, int M, int N);
DACE_EXPORTED void __dace_exit_cuda(gemm_t *__state);



int __dace_init_cuda(gemm_t *__state, int K, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    __state->gpu_context = new dace::cuda::Context(2, 4);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 4; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(gemm_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 4; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void initialize_matmul_result_1_0_1(float * __restrict__ output, int M, int N) {
    constexpr long long VECLEN = 4;
    constexpr long long size_thread_block_tile_m = 128;
    constexpr long long size_thread_block_tile_n = 128;
    constexpr long long num_thread_blocks_m = 32;
    constexpr long long num_thread_blocks_n = 32;
    constexpr long long num_warps_n = 2;
    constexpr long long num_K_tiles = 512;
    constexpr long long size_warp_tile_m = 32;
    constexpr long long size_warp_tile_n = 64;
    constexpr long long size_thread_tile_m = 8;
    constexpr long long size_thread_tile_n = 8;
    constexpr long long warp_width = 8;
    constexpr long long warp_height = 4;
    constexpr long long size_K_tile = 8;
    constexpr long long size_K_split = 4096;
    constexpr long long SWIZZLE = 1;
    constexpr long long SPLIT_K = 1;
    constexpr long long num_threads_per_threadblock = 256;
    {
        {
            int j = (blockIdx.x * 32 + threadIdx.x);
            int i = (blockIdx.y * 1 + threadIdx.y);
            if (j < N) {
                {
                    {
                        float out;

                        ///////////////////
                        // Tasklet code (matmul_init)
                        out = 0;
                        ///////////////////

                        output[((N * i) + j)] = out;
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_initialize_matmul_result_1_0_1(gemm_t *__state, float * __restrict__ output, int M, int N);
void __dace_runkernel_initialize_matmul_result_1_0_1(gemm_t *__state, float * __restrict__ output, int M, int N)
{

    void  *initialize_matmul_result_1_0_1_args[] = { (void *)&output, (void *)&M, (void *)&N };
    hipLaunchKernel((void*)initialize_matmul_result_1_0_1, dim3(int_ceil(int_ceil(N, 1), 32), int_ceil(int_ceil(M, 1), 1), int_ceil(1, 1)), dim3(32, 1, 1), initialize_matmul_result_1_0_1_args, 0, __state->gpu_context->streams[0]);
}
__global__ void Thread_block_grid_1_1_9(const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N) {
    constexpr long long VECLEN = 4;
    constexpr long long size_thread_block_tile_m = 128;
    constexpr long long size_thread_block_tile_n = 128;
    constexpr long long num_thread_blocks_m = 32;
    constexpr long long num_thread_blocks_n = 32;
    constexpr long long num_warps_n = 2;
    constexpr long long num_K_tiles = 512;
    constexpr long long size_warp_tile_m = 32;
    constexpr long long size_warp_tile_n = 64;
    constexpr long long size_thread_tile_m = 8;
    constexpr long long size_thread_tile_n = 8;
    constexpr long long warp_width = 8;
    constexpr long long warp_height = 4;
    constexpr long long size_K_tile = 8;
    constexpr long long size_K_split = 4096;
    constexpr long long SWIZZLE = 1;
    constexpr long long SPLIT_K = 1;
    constexpr long long num_threads_per_threadblock = 256;
    {
        {
            int thread_block_j = blockIdx.x;
            int thread_block_i = blockIdx.y;


    constexpr int M_WARPS = THREADBLOCK_TILE_M / WARP_TILE_M;
	constexpr int N_WARPS = THREADBLOCK_TILE_N / WARP_TILE_N;

	constexpr int N_THREADS = WARP_TILE_N / THREAD_TILE_N;
	constexpr int M_THREADS = WARP_TILE_M / THREAD_TILE_M;

    const int WarpId = threadIdx.x / 32;
	const int threadId = threadIdx.x % 32;

	const int WarpIdx = WarpId % N_WARPS;
	const int WarpIdy = WarpId / N_WARPS;

	int LaneIdx;
	int LaneIdy;

	if (N_THREADS == 1) {

		LaneIdx = 0;
		LaneIdy = threadId;

	} else if (N_THREADS == 2) {

		LaneIdx = (((threadId & 0x60) >> 4) | (threadId & 1));
		LaneIdy = ((threadId >> 1) & (M_THREADS - 1));

	} else if (N_THREADS == 4) {

		LaneIdx = (((threadId & 0x30) >> 3) | (threadId & 1));
		LaneIdy = ((threadId >> 1) & (M_THREADS - 1));

	} else if (N_THREADS == 8) {

		LaneIdx = (((threadId & 0x18) >> 2) | (threadId & 1));
		LaneIdy = ((threadId >> 1) & (M_THREADS - 1));

	} else if (N_THREADS == 16) {

		LaneIdx = (((threadId & 0x1c) >> 1) | (threadId & 1));
		LaneIdy = ((threadId >> 1) & (M_THREADS - 1));

	} else if (N_THREADS == 32) {

		LaneIdx = threadId;
		LaneIdy = 0;
	}
            {
                for (auto k_tile = 0; k_tile < num_K_tiles; k_tile += 1) {
                    __shared__ float shared_memory_A[1024];
                    __shared__ float shared_memory_B[1024];
                    dace::GlobalToShared2D<float, max(1, num_threads_per_threadblock), 1, 1, size_thread_block_tile_m, size_K_tile, 8, 1, true>(input_A + (((K * size_thread_block_tile_m) * thread_block_i) + (k_tile * size_K_tile)), K, 1, shared_memory_A);
                    dace::GlobalToShared2D<float, max(1, num_threads_per_threadblock), 1, 1, size_K_tile, size_thread_block_tile_n, 128, 1, true>(input_B + (((N * k_tile) * size_K_tile) + (size_thread_block_tile_n * thread_block_j)), N, 1, shared_memory_B);
                    {
                        {
                            float register_storage_C[64]  DACE_ALIGN(64) = {0};
                            __syncthreads();
                            int thread = threadIdx.x;
                            if (thread < num_threads_per_threadblock) {
                                {
                                    for (auto k = 0; k < size_K_tile; k += 1) {
                                        float register_storage_A[8]  DACE_ALIGN(64);
                                        float register_storage_B[8]  DACE_ALIGN(64);

                                        dace::CopyND<float, 1, false, size_thread_tile_m>::template ConstDst<1>::Copy(
                                        shared_memory_A + ((k + ((8 * size_thread_tile_m) * ((thread % 32) / warp_width))) + ((8 * size_warp_tile_m) * ((thread / 32) / num_warps_n))), register_storage_A, 8);

                                        dace::CopyND<float, 1, false, size_thread_tile_n>::template ConstDst<1>::Copy(
                                        shared_memory_B + (((128 * k) + (size_thread_tile_n * ((thread % 32) % warp_width))) + (size_warp_tile_n * ((thread / 32) % num_warps_n))), register_storage_B, 1);
                                        {
                                            #pragma unroll
                                            for (auto i = 0; i < size_thread_tile_m; i += 1) {
                                                #pragma unroll
                                                for (auto j = 0; j < size_thread_tile_n; j += 1) {
                                                    {
                                                        float __a = register_storage_A[i];
                                                        float __b = register_storage_B[j];
                                                        float __out;

                                                        ///////////////////
                                                        // Tasklet code (matrix_multiplication)
                                                        __out = (__a * __b);
                                                        ///////////////////

                                                        dace::wcr_fixed<dace::ReductionType::Sum, float>::reduce(register_storage_C + ((8 * i) + j), __out);
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }

                                dace::CopyND<float, 1, false, size_thread_tile_m, size_thread_tile_n>::template ConstSrc<8, 1>::Accumulate_atomic(
                                register_storage_C, output + ((((N * (((size_thread_block_tile_m * thread_block_i) + (size_thread_tile_m * ((thread % 32) / warp_width))) + (size_warp_tile_m * ((thread / 32) / num_warps_n)))) + (size_thread_block_tile_n * thread_block_j)) + (size_thread_tile_n * ((thread % 32) % warp_width))) + (size_warp_tile_n * ((thread / 32) % num_warps_n))), [] (const float& x, const float& y) { return (x + y); }, N, 1);
                            }
                        }
                    }
                    __syncthreads();
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_Thread_block_grid_1_1_9(gemm_t *__state, const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N);
void __dace_runkernel_Thread_block_grid_1_1_9(gemm_t *__state, const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N)
{

    void  *Thread_block_grid_1_1_9_args[] = { (void *)&input_A, (void *)&input_B, (void *)&output, (void *)&K, (void *)&M, (void *)&N };
    hipLaunchKernel((void*)Thread_block_grid_1_1_9, dim3(int_ceil(num_thread_blocks_n, 1), int_ceil(num_thread_blocks_m, 1), 1), dim3(max(1, num_threads_per_threadblock), 1, 1), Thread_block_grid_1_1_9_args, 0, __state->gpu_context->streams[0]);
}

