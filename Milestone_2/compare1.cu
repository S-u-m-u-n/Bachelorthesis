#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>

constexpr long long VECLEN = 4;
constexpr float alpha = 1;
constexpr float beta = 0;
constexpr long long size_thread_block_tile_m = 128;
constexpr long long size_thread_block_tile_n = 128;
constexpr long long size_K_tile = 8;
constexpr long long num_thread_blocks_m = 32;
constexpr long long num_thread_blocks_n = 32;
constexpr long long num_K_tiles = 512;
constexpr long long size_warp_tile_m = 32;
constexpr long long size_warp_tile_n = 64;
constexpr long long size_thread_tile_m = 8;
constexpr long long size_thread_tile_n = 8;
constexpr long long SPLIT_K = 1;
constexpr long long num_threads_per_threadblock = 256;

struct gemm_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(gemm_t *__state, int K, int M, int N);
DACE_EXPORTED void __dace_exit_cuda(gemm_t *__state);



int __dace_init_cuda(gemm_t *__state, int K, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    __state->gpu_context = new dace::cuda::Context(2, 4);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 4; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(gemm_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 4; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void initialize_matmul_result_1_0_1(float * __restrict__ output, int M, int N) {
    constexpr long long VECLEN = 4;
    constexpr long long size_thread_block_tile_m = 128;
    constexpr long long size_thread_block_tile_n = 128;
    constexpr long long num_thread_blocks_m = 32;
    constexpr long long num_thread_blocks_n = 32;
    constexpr long long num_warps_n = 2;
    constexpr long long num_K_tiles = 512;
    constexpr long long size_warp_tile_m = 32;
    constexpr long long size_warp_tile_n = 64;
    constexpr long long size_thread_tile_m = 8;
    constexpr long long size_thread_tile_n = 8;
    constexpr long long warp_width = 8;
    constexpr long long warp_height = 4;
    constexpr long long size_K_tile = 8;
    constexpr long long size_K_split = 4096;
    constexpr long long SWIZZLE = 1;
    constexpr long long SPLIT_K = 1;
    constexpr long long num_threads_per_threadblock = 256;
    {
        {
            int j = (blockIdx.x * 32 + threadIdx.x);
            int i = (blockIdx.y * 1 + threadIdx.y);
            if (j < N) {
                {
                    {
                        float out;

                        ///////////////////
                        // Tasklet code (matmul_init)
                        out = 0;
                        ///////////////////

                        output[((N * i) + j)] = out;
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_initialize_matmul_result_1_0_1(gemm_t *__state, float * __restrict__ output, int M, int N);
void __dace_runkernel_initialize_matmul_result_1_0_1(gemm_t *__state, float * __restrict__ output, int M, int N)
{

    void  *initialize_matmul_result_1_0_1_args[] = { (void *)&output, (void *)&M, (void *)&N };
    hipLaunchKernel((void*)initialize_matmul_result_1_0_1, dim3(int_ceil(int_ceil(N, 1), 32), int_ceil(int_ceil(M, 1), 1), int_ceil(1, 1)), dim3(32, 1, 1), initialize_matmul_result_1_0_1_args, 0, __state->gpu_context->streams[0]);
}
__global__ void Thread_block_grid_1_1_9(const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N) {
    constexpr long long VECLEN = 4;
    constexpr long long size_thread_block_tile_m = 128;
    constexpr long long size_thread_block_tile_n = 128;
    constexpr long long num_thread_blocks_m = 32;
    constexpr long long num_thread_blocks_n = 32;
    constexpr long long num_warps_n = 2;
    constexpr long long num_K_tiles = 512;
    constexpr long long size_warp_tile_m = 32;
    constexpr long long size_warp_tile_n = 64;
    constexpr long long size_thread_tile_m = 8;
    constexpr long long size_thread_tile_n = 8;
    constexpr long long warp_width = 8;
    constexpr long long warp_height = 4;
    constexpr long long size_K_tile = 8;
    constexpr long long size_K_split = 4096;
    constexpr long long SWIZZLE = 1;
    constexpr long long SPLIT_K = 1;
    constexpr long long num_threads_per_threadblock = 256;
    {
        {
            int thread_block_j = blockIdx.x;
            int thread_block_i = blockIdx.y;
            {
                for (auto k_tile = 0; k_tile < num_K_tiles; k_tile += 1) {
                    __shared__ float shared_memory_A[1024];
                    __shared__ float shared_memory_B[1024];
                    dace::GlobalToShared2D<float, max(1, num_threads_per_threadblock), 1, 1, size_thread_block_tile_m, size_K_tile, 8, 1, true>(input_A + (((K * size_thread_block_tile_m) * thread_block_i) + (k_tile * size_K_tile)), K, 1, shared_memory_A);
                    dace::GlobalToShared2D<float, max(1, num_threads_per_threadblock), 1, 1, size_K_tile, size_thread_block_tile_n, 128, 1, true>(input_B + (((N * k_tile) * size_K_tile) + (size_thread_block_tile_n * thread_block_j)), N, 1, shared_memory_B);
                    {
                        {
                            float register_storage_C[64]  DACE_ALIGN(64) = {0};
                            __syncthreads();
                            int thread = threadIdx.x;
                            if (thread < num_threads_per_threadblock) {
                                {
                                    for (auto k = 0; k < size_K_tile; k += 1) {
                                        float register_storage_A[8]  DACE_ALIGN(64);
                                        float register_storage_B[8]  DACE_ALIGN(64);

                                        dace::CopyND<float, 1, false, size_thread_tile_m>::template ConstDst<1>::Copy(
                                        shared_memory_A + ((k + ((8 * size_thread_tile_m) * ((thread % 32) / warp_width))) + ((8 * size_warp_tile_m) * ((thread / 32) / num_warps_n))), register_storage_A, 8);

                                        dace::CopyND<float, 1, false, size_thread_tile_n>::template ConstDst<1>::Copy(
                                        shared_memory_B + (((128 * k) + (size_thread_tile_n * ((thread % 32) % warp_width))) + (size_warp_tile_n * ((thread / 32) % num_warps_n))), register_storage_B, 1);
                                        {
                                            #pragma unroll
                                            for (auto i = 0; i < size_thread_tile_m; i += 1) {
                                                #pragma unroll
                                                for (auto j = 0; j < size_thread_tile_n; j += 1) {
                                                    {
                                                        float __a = register_storage_A[i];
                                                        float __b = register_storage_B[j];
                                                        float __out;

                                                        ///////////////////
                                                        // Tasklet code (matrix_multiplication)
                                                        __out = (__a * __b);
                                                        ///////////////////

                                                        dace::wcr_fixed<dace::ReductionType::Sum, float>::reduce(register_storage_C + ((8 * i) + j), __out);
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }

                                dace::CopyND<float, 1, false, size_thread_tile_m, size_thread_tile_n>::template ConstSrc<8, 1>::Accumulate_atomic(
                                register_storage_C, output + ((((N * (((size_thread_block_tile_m * thread_block_i) + (size_thread_tile_m * ((thread % 32) / warp_width))) + (size_warp_tile_m * ((thread / 32) / num_warps_n)))) + (size_thread_block_tile_n * thread_block_j)) + (size_thread_tile_n * ((thread % 32) % warp_width))) + (size_warp_tile_n * ((thread / 32) % num_warps_n))), [] (const float& x, const float& y) { return (x + y); }, N, 1);
                            }
                        }
                    }
                    __syncthreads();
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_Thread_block_grid_1_1_9(gemm_t *__state, const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N);
void __dace_runkernel_Thread_block_grid_1_1_9(gemm_t *__state, const float * __restrict__ input_A, const float * __restrict__ input_B, float * __restrict__ output, int K, int M, int N)
{

    void  *Thread_block_grid_1_1_9_args[] = { (void *)&input_A, (void *)&input_B, (void *)&output, (void *)&K, (void *)&M, (void *)&N };
    hipLaunchKernel((void*)Thread_block_grid_1_1_9, dim3(int_ceil(num_thread_blocks_n, 1), int_ceil(num_thread_blocks_m, 1), 1), dim3(max(1, num_threads_per_threadblock), 1, 1), Thread_block_grid_1_1_9_args, 0, __state->gpu_context->streams[0]);
}

