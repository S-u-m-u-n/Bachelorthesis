#include "hip/hip_runtime.h"
#include "helper.hpp"
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

const int TILE_SIZE = 32;
const int N = 4096;

// Idea: split the matrices up in several smaller tiles and use shared memory
template <typename T> __global__ void matrix_multiplication_improved(const T *A, const T *B, T *C) {
    // shared memory is private for each thread block
    __shared__ T tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ T tile_B[TILE_SIZE][TILE_SIZE];

    int tid_y = blockIdx.y * blockDim.y + threadIdx.y; // = row
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x; // = column
    T tmp = 0;

    for (int tile = 0; tile < N; tile += TILE_SIZE) {
        // printf("(threadIdx.x, threadIdx.y): (%d, %d)\n", threadIdx.x, threadIdx.y);
        assert(threadIdx.y < TILE_SIZE);
        assert(threadIdx.x < TILE_SIZE);
        // printf("Acessing A[%d]\n", tid_y * N + tile + threadIdx.x);
        // printf("Acessing B[%d]\n", tile * N + threadIdx.y * N + tid_x);
        assert(tid_y * N + tile + threadIdx.x < N * N);
        assert(tile * N + threadIdx.y * N + tid_x < N * N);

        // printf("tile_A[%d][%d] = %f\n", threadIdx.y, threadIdx.x, tile_A[threadIdx.y][threadIdx.x]);
        // printf("A[%d] = %f\n", tid_y * N + tile + threadIdx.x, A[tid_y * N + tile + threadIdx.x]);
        tile_A[threadIdx.y][threadIdx.x] = A[tid_y * N + tile + threadIdx.x];
        tile_B[threadIdx.y][threadIdx.x] = B[tile * N + threadIdx.y * N + tid_x];
        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            tmp += tile_A[threadIdx.y][i] * tile_B[i][threadIdx.x];
        }
        __syncthreads();
    }

    C[tid_y * N + tid_x] = tmp;
}

int main() {
    const size_t memsize = N * N * sizeof(double);
    std::cout << info << "Memory Size per Array: " << memsize << "\n";
    double *A = (double *)malloc(memsize);
    double *B = (double *)malloc(memsize);
    double *C = (double *)malloc(memsize);
    double *dev_A, *dev_B, *dev_C;
    HANDLE_ERROR(hipMalloc((void **)&dev_A, memsize));
    HANDLE_ERROR(hipMalloc((void **)&dev_B, memsize));
    HANDLE_ERROR(hipMalloc((void **)&dev_C, memsize));

    // fill the two matrices with random double values
    fillMatrix(A, N, N);
    // std::cout << info << "A[0] = " << A[0] << ", A[N * N -1] = " << A[N * N - 1] << "\n";
    fillMatrix(B, N, N);
    // std::cout << info << "B[0] = " << B[0] << ", B[N * N -1] = " << B[N * N - 1] << "\n";

    // // print the values of the two matrices
    // printMatrix(A, N, N);
    // printMatrix(B, N, N);

    // Initialize the grid and block dimensions
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
    // Assuming N is a multiple of TILE_SIZE, otherwise we need to add padding or check bounds in kernel
    dim3 dimGrid(N / TILE_SIZE, N / TILE_SIZE, 1);

    // start profiling
    hipProfilerStart();

    HANDLE_ERROR(hipMemcpy(dev_A, A, memsize, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_B, B, memsize, hipMemcpyHostToDevice));

    // std::cout << info << "Launching CUDA kernel...\n";
    matrix_multiplication_improved<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C);
    hipDeviceSynchronize();
    // std::cout << info << "CUDA kernel is finished!\n";

    HANDLE_ERROR(hipMemcpy(C, dev_C, memsize, hipMemcpyDeviceToHost));
    // std::cout << info << "Result copied back to host.\n";

    // stop profiling
    hipProfilerStop();

    HANDLE_ERROR(hipFree(dev_A));
    HANDLE_ERROR(hipFree(dev_B));
    HANDLE_ERROR(hipFree(dev_C));

// printMatrix(C, N, N);
// verify that solution is correct
#ifdef VERIFY
    {
        if (verify(A, B, C, N))
            std::cout << success << "Matrix multiplication successful!" << std::endl;
        else
            std::cout << error << "Matrix multiplication unsuccessful. :(" << std::endl;
    }
#endif

    std::cout << success << "Matrix multiplication successful!" << std::endl;

    free(A);
    free(B);
    free(C);

    return 0;
}

